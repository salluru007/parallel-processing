void rgb2gray_gpu (unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, unsigned int width, unsigned int height){
    Timer timer;

    //Allocate GPU memory
    startTime(&timer);

    unsigned char *red_d, *green_d, *blue_d, *blue_d, *gray_d; 
    hipMalloc((void**) &red_d, width*height* sizeof(unsigned char)); 
    hipMalloc((void**) &green_d, width*height*sizeof(unsigned char)); 
    hipMalloc((void**)&blue_d, width height sizeof(unsigned char)); 
    hipMalloc((void**) &gray_d, width*height*sizeof(unsigned chur)); 
    cudaDevicesynchronize;
    stopTimer(&timer):
    printElapsedTime(timer, "Allocation time");

    //Copy data to GPU
    
    startTime(&timer);
    hipMemcpy(red_d, red, width*height sizeof(unsigned char), hipMemcpyHostToDevice); hipMemcpy(green_d, green, width*height sizeof(unsigned char), hipMemcpyHostToDevice); hipMemcpy(blue_d, blue, width*height sizeof(unsigned char), hipMemcpyHostToDevice); cudaDevicesynchronize();
    stoptime(&timer);
    printelapsedTime(timer, "Copy to GPU time");
    // call kernel
    startTime(&ymer);
    hipDeviceSynchronize();
    stoptime(&timer))
    printelapsedTime(timer, "Kernel time"; GREEN);
    
    // Copy data from GPU startTime(&timer);
}